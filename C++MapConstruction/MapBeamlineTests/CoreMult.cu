/**
 * Copyright 2013 Diana-Andreea Popescu, EPFL, Switzerland.  All rights reserved.
 *
 */

// System includes
#include <stdio.h>
#include <assert.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/remove.h>

#define MAX_EXP	100

inline
void checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
	exit(EXIT_FAILURE);
  }
}

struct is_order_less
{
	__host__ __device__
	bool operator() (const int x)
	{
		return (x < 0);
	}
};

void initPol(unsigned int *exps, unsigned int dim, double *coeffs, unsigned int nvars)
{
    for (unsigned int i = 0; i < dim; ++i)
    {
        for (unsigned int k = 0; k < nvars; ++k)
			exps[i + k * dim] = rand() % 20;
		coeffs[i] = 1;
    }
//	for (unsigned int i = 0; i < dim * nvars; ++i)
//		  printf("%d ", exps[i]);
}

/**
 * Run a multivariate polynomial multiplication using CUDA
 */
int polynomMultiply(int argc, char **argv, int block_size, unsigned int &dimA, unsigned int &dimB, 
		    unsigned int &order, unsigned int &nvars, int n)
{
  //omp_set_num_threads(n);
    // Allocate host memory for polynoms A and B
    unsigned int size_A = dimA * nvars;
    unsigned int mem_size_exp_A = sizeof(unsigned int) * size_A;
    unsigned int mem_size_coeff_A = sizeof(double) * dimA;
    unsigned int *exp_A = (unsigned int*) malloc(mem_size_exp_A);
   
    double *coeff_A = (double*) malloc(mem_size_coeff_A);
   
    unsigned int size_B = dimB * nvars;
    unsigned int mem_size_exp_B = sizeof(unsigned int) * size_B;
    unsigned int mem_size_coeff_B = sizeof(double) * dimB;
    unsigned int *exp_B = (unsigned int*) malloc(mem_size_exp_B);
    double *coeff_B = (double*)malloc(mem_size_coeff_B);

    // Initialize host memory
    initPol(exp_A, dimA, coeff_A, nvars);
    initPol(exp_B, dimB, coeff_B, nvars);

    // Allocate device memory
    double *final_coeff_C;
    unsigned long long *e_keys_C;
    unsigned long long *final_keys_C;

    // Allocate host polynom C
    unsigned int dimC = dimA * dimB;
    unsigned int size_C = dimA * dimB * nvars;
    unsigned int mem_size_exp_C = size_C * sizeof(unsigned int);
    unsigned int *exp_C = (unsigned int*)malloc(mem_size_exp_C);

    unsigned int mem_size_keys_C = dimC * sizeof(unsigned long long); 
    unsigned int mem_size_coeff_C = sizeof(double) * dimC;
    double *coeff_C = (double*)malloc(mem_size_coeff_C);

    final_coeff_C = (double*)malloc(mem_size_coeff_C);

    e_keys_C = (unsigned long long*)malloc(mem_size_keys_C);

    final_keys_C = (unsigned long long*)malloc(mem_size_keys_C);

    //STENCIL FOR TRUNCATION
    int *stencil = NULL;
    unsigned int mem_size_stencil = sizeof(int) * dimC;
    stencil = (int*)malloc(mem_size_stencil);
    
    printf("Computing result ...\n");

    // Execute the kernel
    int nIter = 1;
    unsigned long long ekey = 0, kd = 0;
    unsigned int sum = 0;
    unsigned int cexp = 0;
    for (int it = 0; it < nIter; it++)
    {
      double start1 = omp_get_wtime();
#pragma omp parallel for shared(exp_A, exp_B, exp_C, coeff_A, coeff_B, coeff_C) firstprivate(ekey, cexp, sum) schedule(static)
      for (int i = 0; i < dimB; i ++)
	for (int j = 0; j < dimA; j ++) {
	  coeff_C[i * dimA + j] = coeff_A[j] * coeff_B[i];
	  for (int k = 0; k < nvars; k ++){
	    cexp = exp_A[j + k * dimA] + exp_B[i + k * dimB];
	    exp_C[i * dimA + j + k * dimC] = cexp;
	    ekey = MAX_EXP * ekey + cexp;
	    //sum += cexp;
	  }
	  e_keys_C[i * dimA + j] = ekey;
	
	}
      double end1 = omp_get_wtime();
      printf("%lf\n", 1000 * (end1 - start1));
      	thrust::device_vector<unsigned long long> keys_C_dev(e_keys_C, e_keys_C + dimC);
	thrust::device_vector<double> coeff_C_dev(coeff_C, coeff_C + dimC);
        thrust::device_vector<unsigned long long> final_keys_C_dev(final_keys_C, final_keys_C + dimC);
        thrust::device_vector<double> final_coeff_C_dev(final_coeff_C, final_coeff_C + dimC);
	double start2 = omp_get_wtime();
	thrust::sort_by_key(keys_C_dev.begin(), keys_C_dev.end(), coeff_C_dev.begin());
	thrust::pair<thrust::device_vector<unsigned long long>::iterator, thrust::device_vector<double>::iterator > end;
	end = thrust::reduce_by_key(keys_C_dev.begin(), keys_C_dev.end(), coeff_C_dev.begin(), final_keys_C_dev.begin(), final_coeff_C_dev.begin());
	int sizeC = end.first - final_keys_C_dev.begin();
	double end2 = omp_get_wtime();
	printf("%lf\n", 1000 * (end2 - start2)); 
	double start3 = omp_get_wtime();
#pragma omp parallel for private(kd, ekey) shared(exp_C, final_keys_C_dev, sizeC) schedule(static)
	for (int i = 0; i < sizeC; i ++){
		ekey = final_keys_C_dev[i];
		for (int k = nvars - 1; k >= 0; k--) {
		  	kd = ekey/MAX_EXP;
			exp_C[i + k * dimC] = ekey - kd * MAX_EXP; 
			ekey = kd;
		}
	}
	double end3 = omp_get_wtime();
	printf("%lf\n", 1000 * (end3 - start3));

	
	/*	thrust::device_ptr<int> stencil_dev(stencil);
		thrust::device_ptr<double> end_coeffs_dev = thrust::remove_if(coeffs_C_dev, coeffs_C_dev + dimC, stencil_dev, is_order_less());
		thrust::device_ptr<unsigned long long> end_keys_dev = thrust::remove_if(keys_C_dev, keys_C_dev + dimC, stencil_dev, is_order_less());
		thrust::sort_by_key(keys_C_dev, end_keys_dev, coeffs_C_dev);
		thrust::pair<thrust::device_ptr<unsigned long long>, thrust::device_ptr<double>> end;
		end = thrust::reduce_by_key(keys_C_dev, end_keys_dev, coeffs_C_dev, final_keys_C_dev, final_coeff_C_dev); 
		getExponentsFromKeysCUDA<1024,6,100><<< grid_exp, threads_exp >>>(e_C, final_keys_C, sizeC);
	//	thrust::sort_by_key(keys_C_dev, keys_C_dev + dimC, values_C_dev);
		
	*/
	
    }



    printf("Checking computed result for correctness: ");
    bool correct = true;

/*	for (int i = 0; i < sizeC; ++i){
		for (int k = 0; k < nvars; ++k){
			printf("%d ", exp_C[i + k * sizeC]);
		}
		printf("%lf \n", coeff_C[i]);
	} */
/*    for (unsigned int i = 0; i < dimA; ++i)
    {
    	for (unsigned int j = 0; j < dimB; ++j)
	{
		double coef = coeff_A[i] * coeff_B[j];
	       	for (unsigned int k = 0; k < nvars; ++k)
		{
			unsigned int expc = exp_A[i + k * dimA] + exp_B[j + k * dimB];
			//printf(" %d ", expc);
			if (expc != exp_C[j * dimA + i + k * dimC])
			{
				printf("Error! Pol dif %d - %d %d\n", i, expc, exp_C[j * dimA + i + k * dimC]);
				correct = false;
			}
		}
       	}
	}*/ 

    printf("%s\n", correct ? "OK" : "FAIL");

    // Clean up memory
    free(exp_A);
    free(exp_B);
    free(exp_C);
    free(coeff_A);
    free(coeff_B);
    free(coeff_C);

    free(e_keys_C);
    free(final_keys_C);
    free(final_coeff_C);
    free(stencil);

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    } 
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Multivariate Polynomial Multiplication Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -nA=NumberOfTermsA (Number of terms of polynom A)\n");
        printf("      -nB=NumberOfTermsB (Number of terms of polynom B)\n");
        printf("	  -x=vars (Number of variables)\n");
		printf("	  -o=order (Order of polynoms).\n");
		printf("	  -b=block_size (Block size).\n");

        exit(EXIT_SUCCESS);
    }

 
    // Use a larger block size for Fermi and above
    int block_size = 16;

    unsigned int dimA = 16 * block_size;
    unsigned int dimB = 8 * block_size;

    // number of terms of polynom A
    if (checkCmdLineFlag(argc, (const char **)argv, "nA"))
    {
        dimA = getCmdLineArgumentInt(argc, (const char **)argv, "nA");
    }

    // number of terms of polynom B
    if (checkCmdLineFlag(argc, (const char **)argv, "nB"))
    {
        dimB = getCmdLineArgumentInt(argc, (const char **)argv, "nB");
    }

	unsigned int order = 6;
	// Order of polynoms
    if (checkCmdLineFlag(argc, (const char **)argv, "o"))
    {
        order = getCmdLineArgumentInt(argc, (const char **)argv, "o");
    }

    int n = 1;
    if (checkCmdLineFlag(argc, (const char **)argv, "n"))
    {
        n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
    }

	unsigned int nvars = 6;
	// Number of variables
    if (checkCmdLineFlag(argc, (const char **)argv, "x"))
    {
		nvars = getCmdLineArgumentInt(argc, (const char **)argv, "x");
    }


    printf("PolynomA(%d), PolynomB(%d), Order = %d, Number of Variables = %d\n", dimA, dimB, order, nvars);

    int polynom_result = polynomMultiply(argc, argv, block_size, dimA, dimB, order, nvars, n);

    exit(polynom_result);
}
